// src/heart.cu


#include <hip/hip_runtime.h>
#include <iostream>

__global__ void hello_from_gpu() {
    printf("Hello from GPU thread %d!\n", threadIdx.x);
}

void launch_cuda_kernel() {
    hello_from_gpu<<<1, 5>>>();
    hipDeviceSynchronize();
}
